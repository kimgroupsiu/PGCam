#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */


#ifndef _VectorAdd_KERNEL_H_
#define _VectorAdd_KERNEL_H_

#include "cuda_kernels.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void copyfloat2uchar(unsigned char *dst, const float *src, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        dst[i] = (unsigned char)src[i];
    }
}

__global__ void copyuchar2float(float *dst, const unsigned char *src, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        dst[i] = (float)src[i];
    }
}

// Device code
extern "C" void VecAdd_kernelGPU(
    float *d_Src1,
    float *d_Src2,
    float *d_Dst,
    int numElements
)
{
    // Launch the Vector Add CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_Src1, d_Src2, d_Dst, numElements);
    //hipError_t err = hipGetLastError();
}

extern "C" void copyfloat2uchar_gpu(
    unsigned char * d_dst,
    float * d_Src,
    int numElements
)
{
    // Launch the Vector Add CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    copyfloat2uchar<<<blocksPerGrid, threadsPerBlock>>>(d_dst, d_Src, numElements);
    //hipError_t err = hipGetLastError();
}

extern "C" void copyuchar2float_gpu(
    float * d_dst,
    unsigned char * d_Src,
    int numElements
)
{
    // Launch the Vector Add CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    copyuchar2float<<<blocksPerGrid, threadsPerBlock>>>(d_dst, d_Src, numElements);
    //hipError_t err = hipGetLastError();
}


#endif
